#include "hip/hip_runtime.h"
/***********************************************************************
 * sobel-gpu-bmtalotta-cs441.cu
 *
 * Implements a Sobel filter on the image that is hard-coded in main.
 * You might add the image name as a command line option if you were
 * to use this more than as a one-off assignment.
 *
 * See https://stackoverflow.com/questions/17815687/image-processing-implementing-sobel-filter
 * or https://blog.saush.com/2011/04/20/edge-detection-with-the-sobel-operator-in-ruby/
 * for info on how the filter is implemented.
 *
 * Compile/run with:  nvcc sobel-cpu.cu -lfreeimage
 *
 ***********************************************************************/
#include "FreeImage.h"
#include "stdio.h"
#include "math.h"

// Returns the index into the 1d pixel array
// Given te desired x,y, and image width
__device__ int pixelIndex(int x, int y, int width)
{
  return (y*width + x);
}

// Returns the sobel value for pixel x,y
__global__ void sobel(int width, int height, char *pixels, int *arr)
{
  //y uses .x and x uses .y for this :(
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  if(x > 0 && y > 0 && x < width - 1 && y < width - 1){
    int x00 = -1;  int x20 = 1;
    int x01 = -2;  int x21 = 2;
    int x02 = -1;  int x22 = 1;
    x00 *= pixels[pixelIndex(x-1,y-1,width)];
    x01 *= pixels[pixelIndex(x-1,y,width)];
    x02 *= pixels[pixelIndex(x-1,y+1,width)];
    x20 *= pixels[pixelIndex(x+1,y-1,width)];
    x21 *= pixels[pixelIndex(x+1,y,width)];
    x22 *= pixels[pixelIndex(x+1,y+1,width)];
    
    int y00 = -1;  int y10 = -2;  int y20 = -1;
    int y02 = 1;  int y12 = 2;  int y22 = 1;
    y00 *= pixels[pixelIndex(x-1,y-1,width)];
    y10 *= pixels[pixelIndex(x,y-1,width)];
    y20 *= pixels[pixelIndex(x+1,y-1,width)];
    y02 *= pixels[pixelIndex(x-1,y+1,width)];
    y12 *= pixels[pixelIndex(x,y+1,width)];
    y22 *= pixels[pixelIndex(x+1,y+1,width)];
    int pixelX = x00 + x01 + x02 + x20 + x21 + x22;
    int pixelY = y00 + y10 + y20 + y02 + y12 + y22;
    arr[pixelIndex(x,y,width)] = (int)sqrtf(pixelX*pixelX +pixelY*pixelY);
  }else{
    arr[pixelIndex(x,y,width)] = pixels[pixelIndex(x,y,width)];
  }
}

int main()
{
  FreeImage_Initialise();
  atexit(FreeImage_DeInitialise);

  // Load image and get the width and height
  FIBITMAP *image;
  image = FreeImage_Load(FIF_PNG, "coins.png", 0);
  if (image == NULL)
  {
      printf("Image Load Problem\n");
      exit(0);
  }
  int imgWidth;
  int imgHeight;
  imgWidth = FreeImage_GetWidth(image);
  imgHeight = FreeImage_GetHeight(image);

  // Convert image into a flat array of chars with the value 0-255 of the
  // greyscale intensity
  int *arr;
  int *dev_arr;
  arr = (int *)malloc(sizeof(int) * imgWidth *imgHeight);
  hipMalloc((void **)&dev_arr, sizeof(int) * imgWidth * imgHeight);

  RGBQUAD aPixel;
  char *pixels;  
  char *dev_pixels;
  int pixIndex = 0;
  pixels = (char *) malloc(sizeof(char)*imgWidth*imgHeight);
  for (int i = 0; i < imgHeight; i++){
    for (int j = 0; j < imgWidth; j++)
    {
      FreeImage_GetPixelColor(image,j,i,&aPixel);
      char grey = ((aPixel.rgbRed + aPixel.rgbGreen + aPixel.rgbBlue)/3);
      pixels[pixIndex++]=grey;
    }
  }
  hipMalloc((void **)&dev_pixels, sizeof(char) * imgWidth *imgHeight);
  hipMemcpy(dev_pixels, pixels, sizeof(char) * imgWidth *imgHeight, hipMemcpyHostToDevice);
  int powOfTwo = 2;
  while ((imgHeight % powOfTwo) == 0 && (imgWidth % powOfTwo) == 0){
    powOfTwo *= 2;
  }
  dim3 numThreadsPerBlock(powOfTwo,powOfTwo);
  int blockX = imgHeight / powOfTwo;
  int blockY = imgWidth / powOfTwo;
  dim3 numBlocks(blockX,blockY);
  // Apply sobel operator to pixels, ignoring the borders
  sobel<<<numBlocks, numThreadsPerBlock>>>(imgWidth, imgHeight, dev_pixels, dev_arr);
  hipMemcpy(arr,dev_arr,sizeof(int) * imgWidth * imgHeight, hipMemcpyDeviceToHost);

  FIBITMAP *bitmap = FreeImage_Allocate(imgWidth, imgHeight, 24);
  for (int i = 1; i < imgWidth-1; i++)
  {
    for (int j = 1; j < imgHeight-1; j++)
    {
    int sVal = arr[j * imgWidth + i];
      aPixel.rgbRed = sVal;
      aPixel.rgbGreen = sVal;
      aPixel.rgbBlue = sVal;
      FreeImage_SetPixelColor(bitmap, i, j, &aPixel);
    }
  }
  FreeImage_Save(FIF_PNG, bitmap, "coins-edge.png", 0);
  free(arr);
  hipFree(dev_arr);
  free(pixels);
  hipFree(dev_pixels);
  FreeImage_Unload(bitmap);
  FreeImage_Unload(image);
  return 0;
}
